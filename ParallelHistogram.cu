#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <dirent.h>
#include <string.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define GRAYLEVELS 256
#define err(msg) {fprintf(stderr, "%s\n", msg); exit(1);}
#define INDIR "./Input/"
#define OUTDIR "./OutputParallel/"

#include "stb_image.h"
#include "stb_image_write.h"
    
DIR * d;
struct dirent * dir;

__global__ void findMin(unsigned int * input, unsigned int * output, int n) {
    extern __shared__ unsigned int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load shared memory with input data, replace zero with UINT_MAX
    sdata[tid] = (i < n) ? (input[i] == 0 ? UINT_MAX : input[i]) : UINT_MAX;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = min(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0)
        output[blockIdx.x] = sdata[0];
    
}

/* Thread/Pixel */
__global__ void calculateHistogram(unsigned char * image, unsigned int * histogram, int imageSize) {
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    if (thid < imageSize)
        atomicAdd(&histogram[image[thid]], 1);
}

__global__ void Scan(unsigned int * input, unsigned int * output, int n) {
    /* Balanced trees - Blelloch (1990) - NVIDIA website chapter 39 */
    extern __shared__ unsigned int temp[]; 
    int thid = threadIdx.x; /* ThreadID */
    int offset = 1;

    temp[2 * thid] = (2 * thid < n) ? input[2 * thid] : 0; 
    temp[2 * thid + 1] = (2 * thid + 1 < n) ? input[2 * thid + 1] : 0; 

    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads(); 
        if (thid < d) { 
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0) { temp[n - 1] = 0; }

    for (int d = 1; d < n; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            unsigned long t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (2 * thid < n) 
        output[2 * thid] = temp[2 * thid];
    if (2 * thid + 1 < n) 
        output[2 * thid + 1] = temp[2 * thid + 1];
}



__device__ unsigned char scale(unsigned long cdf, unsigned long cdfmin, unsigned long imageSize) {
    float scale;
    
    scale = (float)(cdf - cdfmin) / (float)(imageSize - cdfmin);
    
    scale = round(scale * (float)(GRAYLEVELS-1));
    
    return (int)scale;
}

__global__ void transformImage(unsigned char * imageIn, unsigned char * imageOut, int width, int height,
                                     unsigned int * cdf, unsigned int cdfMin, unsigned int imageSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;
        unsigned char pxVal = imageIn[index];
        imageOut[index] = scale(cdf[pxVal], cdfMin, imageSize);
    }
}

unsigned char * fetchImage(int * width, int * height) {
    
    while ((dir = readdir(d)) != NULL) {
        if (strstr(dir->d_name, "jpg") != NULL || strstr(dir->d_name, "png") != NULL)
            break;
    }
    
    /* When out of files so it doesn't give seg fault*/
    if (dir == NULL) 
        return NULL;

    char * imgName = (char *) calloc(strlen(INDIR) + strlen(dir->d_name) + 1, sizeof(char));
    sprintf(imgName, "%s%s", INDIR, dir->d_name);

    int chCount; /* chCount represents number of channels, since its grayscale we need only 1 --> defined as the last arg in stbi_load*/

    unsigned char * imageIn = stbi_load(imgName, width, height, &chCount, 1);
    
    free(imgName);
    return imageIn;
}

int main() {
    d = opendir(INDIR);

    if (d == NULL)
        err("Failed to open dir");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int width, height, imageSize;
    unsigned char * imageIn;
    
    while ((imageIn = fetchImage(&width, &height)) != NULL) {
        imageSize = width * height;

        hipEventRecord(start);

        int threadsPerBlock = 128;
        int blocksPerGrid = (imageSize + threadsPerBlock - 1) / threadsPerBlock;
        int numElements = GRAYLEVELS; 

        unsigned int * histogram = (unsigned int *) calloc(numElements, sizeof(unsigned int));
        unsigned int * cdf = (unsigned int *) malloc(numElements * sizeof(unsigned int));
        unsigned char * imageOut = (unsigned char *) malloc(imageSize * sizeof(unsigned char));

        unsigned char * d_imageIn, * d_imageOut;
        unsigned int * d_histogram, * d_cdf;
        unsigned int * d_intermediate, * h_intermediate;
        
        h_intermediate = (unsigned int *) malloc(blocksPerGrid * sizeof(unsigned int));

        hipMalloc(&d_intermediate, blocksPerGrid * sizeof(unsigned int));
        hipMalloc(&d_imageIn, imageSize * sizeof(unsigned char));
        hipMalloc(&d_imageOut, imageSize * sizeof(unsigned char));
        hipMalloc(&d_histogram, numElements * sizeof(unsigned int));
        hipMalloc(&d_cdf, numElements * sizeof(unsigned int));
        
        hipMemset(d_histogram, 0, numElements * sizeof(unsigned int));
        hipMemset(d_cdf, 0, numElements * sizeof(unsigned int));

        hipMemcpy(d_imageIn, imageIn, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);

        calculateHistogram<<<blocksPerGrid, threadsPerBlock>>>(d_imageIn, d_histogram, imageSize);
        hipMemcpy(histogram, d_histogram, numElements * sizeof(unsigned int), hipMemcpyDeviceToHost);

        Scan<<<1, numElements / 2, numElements * sizeof(unsigned int)>>>(d_histogram, d_cdf, numElements);
        hipMemcpy(cdf, d_cdf, numElements * sizeof(unsigned int), hipMemcpyDeviceToHost);

        findMin<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(unsigned int)>>>(d_cdf, d_intermediate, numElements);
        hipMemcpy(h_intermediate, d_intermediate, blocksPerGrid * sizeof(unsigned int), hipMemcpyDeviceToHost);

        unsigned int minNonZero = UINT_MAX;
        for (int i = 0; i < blocksPerGrid; i++)
            if (h_intermediate[i] < minNonZero)
                minNonZero = h_intermediate[i];


        unsigned int cdfMin = (minNonZero == UINT_MAX) ? 0 : minNonZero;

        dim3 threads(16, 16);
        dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
        transformImage<<<blocks, threads>>>(d_imageIn, d_imageOut, width, height, d_cdf, cdfMin, imageSize);

        hipMemcpy(imageOut, d_imageOut, imageSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

        char * name = (char *) calloc(1000, sizeof(char));
        sprintf(name, "%s%s", OUTDIR, dir->d_name);
        stbi_write_png(name, width, height, 1, imageOut, width);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        printf("Image: %s width:%d height:%d processing time: %f ms\n", dir->d_name, width, height, ms);

        hipFree(d_imageIn);
        hipFree(d_imageOut);
        hipFree(d_histogram);
        hipFree(d_cdf);
        free(name);
        free(imageIn);
        free(imageOut);
        free(histogram);
        free(cdf);
    }
    closedir(d);

    return 0;
}